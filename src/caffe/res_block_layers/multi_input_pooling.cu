#include "hip/hip_runtime.h"
/*!
 *  \brief     A helper class for {@link MultiStageMeanfieldLayer} class, which is the Caffe layer that implements the
 *             CRF-RNN described in the paper: Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             This class itself is not a proper Caffe layer although it behaves like one to some degree.
 *
 *  \authors   Sadeep Jayasumana, Bernardino Romera-Paredes, Shuai Zheng, Zhizhong Su.
 *  \version   1.0
 *  \date      2015
 *  \copyright Torr Vision Group, University of Oxford.
 *  \details   If you use this code, please consider citing the paper:
 *             Shuai Zheng, Sadeep Jayasumana, Bernardino Romera-Paredes, Vibhav Vineet, Zhizhong Su, Dalong Du,
 *             Chang Huang, Philip H. S. Torr. Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             For more information about CRF-RNN, please visit the project website http://crfasrnn.torr.vision.
 */
#include <vector>
#include <math.h>
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/loss_layer.hpp"
#include "caffe/layers/softmax_layer.hpp"
#include "caffe/res_block_layers/multi_input_pooling_layer.hpp"


namespace caffe {

template <typename Dtype>
__global__ void pooling_forward_kernel(int nthreads, Dtype** bottom, Dtype* top, int bottomN) {
    CUDA_KERNEL_LOOP(index, nthreads){
        
        Dtype max_value = 1e-8;
        for(int i=0; i<bottomN; i++)
        {
            Dtype temp_value = bottom[i][index];
            if(temp_value > max_value) max_value = temp_value;
        }
        top[index] = max_value;
    }
}


template <typename Dtype>
__global__ void pooling_backward_kernel(int nthreads,const Dtype* top_diff,
                                        Dtype ** bottom_data,
                                        Dtype ** bottom_diff,int bottomN) {
    CUDA_KERNEL_LOOP(index, nthreads){
        
        Dtype max_value = 1e-8;
        int max_Index = 0;
        for(int i=0; i<bottomN; i++)
        {
            Dtype temp_value = bottom_data[i][index];
            if(temp_value > max_value) {
                max_value = temp_value;
                max_Index = i;
            }
        }
        bottom_diff[max_Index][index] = top_diff[index];
    }
}

template <typename Dtype>
void MultiInputPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                             const vector<Blob<Dtype>*>& top)
{
    Dtype ** bottom_data = (Dtype **)malloc(sizeof(Dtype*)*bottom.size());
    for(int i=0;i<bottom.size(); i++)
    {
        bottom_data[i] = (Dtype *) bottom[i]->gpu_data();
    }

    Dtype * top_data=top[0]->mutable_gpu_data();
    
    int count = bottom[0]->count();
    pooling_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, bottom_data, top_data,  bottom.size());
    
    free(bottom_data);
}
template <typename Dtype>
void MultiInputPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                               const vector<bool>& propagate_down,
                                               const vector<Blob<Dtype>*>& bottom)
{
    Dtype ** bottom_data = (Dtype **)malloc(sizeof(Dtype*)*bottom.size());
    for(int i=0;i<bottom.size(); i++)
    {
        bottom_data[i] = (Dtype *) bottom[i]->gpu_data();
    }
    
    Dtype ** bottom_diff = (Dtype **)malloc(sizeof(Dtype*)*bottom.size());
    for(int i=0;i<bottom.size(); i++)
    {
        bottom_diff[i] = bottom[i]->mutable_gpu_diff();
    }
    
    Dtype * top_diff=(Dtype *) top[0]->gpu_diff();
    
    int count = bottom[0]->count();
    pooling_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
    (count, top_diff,  bottom_data, bottom_diff, bottom.size());
    
    free(bottom_data);
    free(bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(MultiInputPoolingLayer);
}  // namespace caffe
