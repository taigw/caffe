#include "hip/hip_runtime.h"
/*!
 *  \brief     A helper class for {@link MultiStageMeanfieldLayer} class, which is the Caffe layer that implements the
 *             CRF-RNN described in the paper: Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             This class itself is not a proper Caffe layer although it behaves like one to some degree.
 *
 *  \authors   Sadeep Jayasumana, Bernardino Romera-Paredes, Shuai Zheng, Zhizhong Su.
 *  \version   1.0
 *  \date      2015
 *  \copyright Torr Vision Group, University of Oxford.
 *  \details   If you use this code, please consider citing the paper:
 *             Shuai Zheng, Sadeep Jayasumana, Bernardino Romera-Paredes, Vibhav Vineet, Zhizhong Su, Dalong Du,
 *             Chang Huang, Philip H. S. Torr. Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             For more information about CRF-RNN, please visit the project website http://crfasrnn.torr.vision.
 */
#include <vector>
#include <math.h>
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/loss_layer.hpp"
#include "caffe/layers/softmax_layer.hpp"
#include "caffe/crf_layers/feature_normalization_layer.hpp"
#include "caffe/crf_layers/pixel_access.hpp"
#include "pixel_access.cu"
namespace caffe {
    
template <typename Dtype>
__global__ void normalization_forward_kernel(const int nthreads, const Dtype* bottom_data, Dtype* top_data,int N, int C, int H, int W) {
    CUDA_KERNEL_LOOP(index, nthreads){
    const int w = index % W;
    const int h = (index/W) % H;
    const int n = (index / W / H);

    Dtype sum = 0;
    for( int c=0; c<C; c++)
    {
        sum +=get_gpu_pixel(bottom_data, N, C, H, W, n, c, h, w);
    }
    if(sum!=0)
    {
        for( int c=0; c<C; c++)
        {
            Dtype temp_value = get_gpu_pixel(bottom_data, N, C, H, W, n, c, h, w)/sum;
            set_gpu_pixel(top_data, N, C, H, W, n, c, h, w, temp_value);
        }
    }
}
}

template <typename Dtype>
__global__ void normalization_backward_kernel(const int nthreads, const Dtype* bottom_data, const Dtype* top_diff, Dtype * bottom_diff, int N, int C, int H, int W) {
    CUDA_KERNEL_LOOP(index, nthreads){
    const int w = index % W;
    const int h = (index/W) % H;
    const int n = (index / W / H);
    
    Dtype sum = 0;
    for( int c=0; c<C; c++)
    {
        sum +=get_gpu_pixel(bottom_data, N, C, H, W, n, c, h, w);
    }
    if(sum!=0)
    {
        for( int c=0; c<C; c++)
        {
            Dtype temp_diff = get_gpu_pixel(top_diff, N, C, H, W, n, c, h, w)/sum;
            set_gpu_pixel(bottom_diff, N, C, H, W, n, c, h, w, temp_diff);
        }
    }
}
}
    


template <typename Dtype>
void FeatureNormalizationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top)
{
    const Dtype * bottom_data  = bottom[0]->gpu_data();
    Dtype * top_data = top[0]->mutable_gpu_data();
    CHECK_EQ(bottom[0]->count(), top[0]->count())<<
        ("input image and output image shoud have the same size");
    
    int count = num_ * height_ * width_;
    normalization_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
    (count, bottom_data, top_data, num_, channels_, height_, width_);
}

template <typename Dtype>
void FeatureNormalizationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom)
{
    if (!propagate_down[0]) {
        return;
    }
    //LOG(INFO) << ("message pasing backward_gpu start.");
    const Dtype * top_diff = top[0]->gpu_diff();
    const Dtype * bottom_data = bottom[0]->gpu_data();
    Dtype * bottom_diff = bottom[0]->mutable_gpu_diff();
    
    int count = num_ * height_ * width_;
    normalization_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
    (count, bottom_data, top_diff, bottom_diff, num_, channels_, height_, width_);
}

INSTANTIATE_LAYER_GPU_FUNCS(FeatureNormalizationLayer);
}  // namespace caffe
