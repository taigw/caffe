#include "hip/hip_runtime.h"
/*!
 *  \brief     A helper class for {@link MultiStageMeanfieldLayer} class, which is the Caffe layer that implements the
 *             CRF-RNN described in the paper: Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             This class itself is not a proper Caffe layer although it behaves like one to some degree.
 *
 *  \authors   Sadeep Jayasumana, Bernardino Romera-Paredes, Shuai Zheng, Zhizhong Su.
 *  \version   1.0
 *  \date      2015
 *  \copyright Torr Vision Group, University of Oxford.
 *  \details   If you use this code, please consider citing the paper:
 *             Shuai Zheng, Sadeep Jayasumana, Bernardino Romera-Paredes, Vibhav Vineet, Zhizhong Su, Dalong Du,
 *             Chang Huang, Philip H. S. Torr. Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             For more information about CRF-RNN, please visit the project website http://crfasrnn.torr.vision.
 */
#include <vector>
#include <math.h>
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/loss_layer.hpp"
#include "caffe/layers/softmax_layer.hpp"
#include "caffe/crf_layers/pairwise_feature_layer.hpp"
#include "caffe/crf_layers/pixel_access.hpp"
#include "pixel_access.cu"

namespace caffe {

template <typename Dtype>
__global__ void feature_forward_kernel(int nthreads, const Dtype* bottom, Dtype* top,
                            int N, int C, int H, int W, int featureN, int neighN) {
    CUDA_KERNEL_LOOP(index, nthreads){
        const int w = index % W;
        const int h = (index / W) % H;
        const int n   = index / W / H;
        // bottom size C, H, W
        // top size C+1, neighN, H*W

        int outC = featureN+1;
        int kernel_size = sqrt(double(neighN+1));
        int kr = (kernel_size -1)/2;
        int h_index = 0;
        int w_index = h*W + w;

        for(int i=-kr; i<=kr; i++)
        {
            for(int j=-kr; j<=kr; j++)
            {
                if(i==0 && j==0)continue;
                Dtype value_diff, p_value, q_value;
                for(int c=0; c<outC; c++)
                {
                    if(c<outC-1)
                    {
                        p_value = get_gpu_pixel(bottom, N, C, H, W, n, c, h, w);
                        q_value = get_gpu_pixel(bottom, N, C, H, W, n, c, i+h, j+w);
                        value_diff=(p_value-q_value);
                    }
                    else
                    {
                        value_diff= sqrt(double(i*i + j*j));
                    }
                    set_gpu_pixel(top, N, outC, neighN, H*W, n, c, h_index, w_index, value_diff);
                }
                h_index++;
            }
        }
    }
}


//template <typename Dtype>
//__global__ void feature_backward_kernel(const Dtype* top_diff,
//                                        const Dtype* bottom_data,
//                                        Dtype * bottom_diff,
//                                        int C, int H, int W, int neighN) {
//    // bottom size C, H, W
//    // top size C+1, neighN, H*W
//    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
//    uint j = (blockIdx.y * blockDim.y) + threadIdx.y;
//    if( i>= H || j>=W) return;
//    int kernel_size = sqrt(double(neighN+1));
//    int kr = (kernel_size -1)/2;
//    
//    for(int c=0; c<C; c++)
//    {
//        int p_index = i*W+j;
//        int q_index = 0;
//        Dtype p_diff = 0;
//        for(int m=-kr; m<=kr; m++)
//        {
//            for(int n=-kr; n<=kr; n++)
//            {
//                if(m==0 && n==0) continue;
//                Dtype t_diff_p, t_diff_q, p_value, q_value;
//            
//                p_value = get_gpu_pixel(bottom_data, C, H, W, c, i, j);
//                q_value = get_gpu_pixel(bottom_data, C, H, W, c, i+m, j+n);
//                t_diff_p = get_gpu_pixel(top_diff, C+1, neighN, H*W, c, q_index, p_index);
//            
//                if(i+m>=0 && i+m<H && j+n>=0 && j+n<W)
//                {
//                    int np_index = (i+m)*W + (j+n);
//                    int nq_index = neighN -1 -q_index;
//                    t_diff_q = get_gpu_pixel(top_diff, C+1, neighN, H*W, c, nq_index, np_index);
//                }
//                else
//                {
//                    t_diff_q = 0;
//                }
//                p_diff += (t_diff_p + t_diff_q)* 2 *(p_value - q_value);
//                q_index++;
//            }
//        }
//        set_gpu_pixel(bottom_diff, C, H, W, c, i, j, p_diff);
//    }
//}

template <typename Dtype>
void feature_backward(const Dtype* top_diff, const Dtype* bottom_data, Dtype * bottom_diff,
                     int N, int C, int H, int W, int neighN) {
//    const Dtype * 
//    dim3 threadsPerBlock(16, 16);
//    dim3 numBlocks((H+threadsPerBlock.x-1)/threadsPerBlock.x,
//                   (W+threadsPerBlock.y-1)/threadsPerBlock.y);
//    long bottom_offset = C*H*W;
//    long top_offset = (C+1)*neighN*H*W;;
//    for(int i=0;i<N; i++)
//    {
//        feature_backward_kernel<Dtype><<<numBlocks, threadsPerBlock>>>(
//            top_diff + i*top_offset,
//            bottom_data + i*bottom_offset,
//            bottom_diff + i*bottom_offset,
//            C, H, W, neighN);
//    }
}
template <typename Dtype>
void PairwiseFeatureLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                             const vector<Blob<Dtype>*>& top)
{
    const Dtype * input_data  = bottom[0]->gpu_data();
    Dtype * output_data=top[0]->mutable_gpu_data();
    CHECK(top[0]->channels() == featureN_ + 1)<<
    ("pairwise feature channel does not match output shape");
    
    CHECK(bottom[0]->height() * bottom[0]->width() == top[0]->width())<<
    ("input image and kernel shoud have matched pixel number")<< bottom[0]->height() <<" "<<bottom[0]->width()<<" "<<top[0]->width();
    
    int count = bottom[0]->num() * bottom[0]->height() * bottom[0]->width();
    feature_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, input_data, output_data,bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(), featureN_, top[0]->height());
//    Forward_cpu(bottom, top);
}
template <typename Dtype>
void PairwiseFeatureLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                               const vector<bool>& propagate_down,
                                               const vector<Blob<Dtype>*>& bottom)
{
//    const Dtype * top_diff = top[0]->gpu_diff();
//    const Dtype * bottom_data = bottom[0]->gpu_data();
//    Dtype * bottom_diff =  bottom[0]->mutable_gpu_diff();
//    feature_backward(top_diff, bottom_data, bottom_diff,
//                     bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
//                     top[0]->height());
}

INSTANTIATE_LAYER_GPU_FUNCS(PairwiseFeatureLayer);
}  // namespace caffe
