#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/crf_layers/resampling_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "pixel_access.cu"

namespace caffe {
    
    
template <typename Dtype>
__global__ void resample_forward_kernel(const int nthreads, const Dtype* data, Dtype* sampled_data,
        int N, int C, int H, int W, int sH, int sW, float sample_rate)
{
    CUDA_KERNEL_LOOP(index, nthreads){
        const int w = index % sW;
        const int h = (index/sW) % sH;
        const int c = (index / sW / sH) % C;
        const int n = index / sW / sH / C;

        float hy = h*sample_rate;
        float wx = w*sample_rate;
        if(hy > H -1 ) hy = H - 1;
        if(wx > W -1) wx = W -1;
        
        int h1 = floor(hy);
        int h2 = ceil(hy);
        int w1 = floor(wx);
        int w2 = ceil(wx);
        float x = (w2-w1>0)? (wx-w1)/(w2-w1) : 0;
        float y = (h2-h1>0)? (hy-h1)/(h2-h1) : 0;
        Dtype Q11 = get_gpu_pixel(data, N, C, H, W, n, c, h1, w1);
        Dtype Q12 = get_gpu_pixel(data, N, C, H, W, n, c, h2, w1);
        Dtype Q21 = get_gpu_pixel(data, N, C, H, W, n, c, h1, w2);
        Dtype Q22 = get_gpu_pixel(data, N, C, H, W, n, c, h2, w2);
        Dtype value = Q11*(1-x)*(1-y) + Q12*(1-x)*y +
            Q21*x*(1-y) + Q22*x*y;
        set_gpu_pixel(sampled_data, N, C, sH, sW, n, c, h, w, value);
    }
}
    
template <typename Dtype>
void ResamplingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top) {
    const Dtype * bottom_data = bottom[0]->gpu_data();
    Dtype * top_data = top[0]->mutable_gpu_data();
    int count = top[0]->count();
    resample_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
    (count, bottom_data, top_data, num_, channels_, height_, width_,
        sampled_height_, sampled_width_,  sample_rate_);
}
    
template <typename Dtype>
__global__ void resample_backward_kernel(const int nthreads, const Dtype* top_diff,
     Dtype* bottom_diff,int N, int C, int sH, int sW, int H, int W, float sample_rate)
{
    CUDA_KERNEL_LOOP(index, nthreads){
        const int bw = index % W;
        const int bh = (index/W) % H;
        const int c = (index / W / H) % C;
        const int n = index / W / H / C;

        int th = bh/sample_rate;
        int tw = bw/sample_rate;
        int th1 = floor((bh-1)/sample_rate);
        int th2 = ceil((bh+1)/sample_rate);
        if(th1<0) th1 = 0;
        if( th2 > sH-1) th2 = sH-1;
        
        int tw1 = floor((bw-1)/sample_rate);
        int tw2 = ceil((bw+1)/sample_rate);
        if(tw1<0) tw1=0;
        if(tw2 > sW-1) tw2 = sW-1;

        Dtype sum_diff = 0.0;
        for(int thIdx = th1+1; thIdx< th2; thIdx++)
        {
            for(int twIdx = tw1+1; twIdx< tw2; twIdx++)
            {
                Dtype t_diff_value = get_gpu_pixel(top_diff, N, C, sH, sW, n, c, thIdx, twIdx);
                
                float hy = thIdx*sample_rate;
                float wx = twIdx*sample_rate;
                if(hy > H -1 ) hy = H - 1;
                if(wx > W -1) wx = W -1;

                int h1 = floor(hy);
                int h2 = ceil(hy);
                int w1 = floor(wx);
                int w2 = ceil(wx);
                float x = (w2-w1>0)? (wx-w1)/(w2-w1) : 0;
                float y = (h2-h1>0)? (hy-h1)/(h2-h1) : 0;

                Dtype diff = 0.0;
                if(thIdx<= th)
                {
                    diff = (twIdx <= tw)? (x*y*t_diff_value) : ((1.0-x)*y*t_diff_value);
                }
                else
                {
                    diff = (twIdx <= tw)? (x*(1.0-y)*t_diff_value) : ((1.0-x)*(1.0-y)*t_diff_value);
                }
                sum_diff += diff;
            }
        }
        set_gpu_pixel(bottom_diff, N, C, H, W, n, c, bh, bw, sum_diff);
    }
}

template <typename Dtype>
void ResamplingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
        return;
    }
    const Dtype * top_diff = top[0]->gpu_diff();
    Dtype * bottom_diff = bottom[0]->mutable_gpu_diff();
    int count = bottom[0]->count();
    resample_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
        (count, top_diff, bottom_diff, num_, channels_, sampled_height_,
         sampled_width_, height_, width_, sample_rate_);
}


INSTANTIATE_LAYER_GPU_FUNCS(ResamplingLayer);


} // namespace caffe