#include "hip/hip_runtime.h"
/*!
 *  \brief     A helper class for {@link MultiStageMeanfieldLayer} class, which is the Caffe layer that implements the
 *             CRF-RNN described in the paper: Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             This class itself is not a proper Caffe layer although it behaves like one to some degree.
 *
 *  \authors   Sadeep Jayasumana, Bernardino Romera-Paredes, Shuai Zheng, Zhizhong Su.
 *  \version   1.0
 *  \date      2015
 *  \copyright Torr Vision Group, University of Oxford.
 *  \details   If you use this code, please consider citing the paper:
 *             Shuai Zheng, Sadeep Jayasumana, Bernardino Romera-Paredes, Vibhav Vineet, Zhizhong Su, Dalong Du,
 *             Chang Huang, Philip H. S. Torr. Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             For more information about CRF-RNN, please visit the project website http://crfasrnn.torr.vision.
 */


#include <vector>
#include <math.h>
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/loss_layer.hpp"
#include "caffe/layers/softmax_layer.hpp"
#include "caffe/crf_layers/message_passing_layer.hpp"
#include "caffe/crf_layers/pixel_access.hpp"
#include "caffe/crf_layers/pairwise_function_intensity_gaussian_layer.hpp"
#include "pixel_access.cu"
namespace caffe {

template <typename Dtype>
__global__ void gaussian_function_kernel(const int nthreads, const Dtype* bottom_data, Dtype * top_data,
                                          int N, int C, int H, int W,
                                         float w1, float theta_beta)
{
    CUDA_KERNEL_LOOP(index, nthreads){
        const int w = index % W;
        const int h = (index/W) % H;
        const int n = index / W / H ;
        
        Dtype isq=0;
        Dtype dsq;
        for(int c=0; c<C; c++)
        {
            Dtype p_value = get_gpu_pixel(bottom_data, N, C, H, W, n, c, h, w);
            if(c<C-1){
                isq += p_value*p_value;
            }
            else{
                dsq = p_value;//*p_value;
            }
        }
//        Dtype p_term = dsq/(2 * theta_alpha * theta_alpha);
//        Dtype i_term = isq/(2 * theta_beta * theta_beta);
//        Dtype bilateral = exp( - i_term - p_term);
//        Dtype spatial = exp(- dsq/(2 * theta_gamma * theta_gamma));
//        Dtype pair_potential = w1*bilateral + w2* spatial;
        
        Dtype i_term = isq/(2 * (C-1) * theta_beta * theta_beta);
        Dtype i_exp = exp(-i_term);
        Dtype pair_potential = (w1/dsq)*i_exp;
        set_gpu_pixel(top_data, N, 1, H, W, n, 0, h, w, pair_potential);
    }
}

template <typename Dtype>
void PairwiseFunctionIntensityGaussianLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                             const vector<Blob<Dtype>*>& top)
{
    //Forward_cpu(bottom, top);
    const Dtype * input_data  = bottom[0]->gpu_data();
//    const Dtype * param_data = param_blob_->gpu_data();
    Dtype * output_data=top[0]->mutable_gpu_data();

//    const Dtype * param_datacpu = param_blob_->cpu_data();
//    LOG(INFO)<<"params: "<<param_datacpu[0]<<" "<<param_datacpu[1]<<" "<<param_datacpu[2]<<" "<<param_datacpu[3]<<" "<<param_datacpu[4];
//
//    const Dtype * param_diff = param_blob_->cpu_diff();
//    LOG(INFO)<<"params diff: "<<param_diff[0]<<" "<<param_diff[1]<<" "<<param_diff[2]<<" "<<param_diff[3]<<" "<<param_diff[4];
    

    CHECK(bottom[0]->height() == top[0]->height() && bottom[0]->width() == top[0]->width() &&
          bottom[0]->num() == top[0]->num() )<<
    ("input size and output size does not match");
    CHECK(top[0]->channels() ==1 )<<
    ("number of output channel should be 1");
    
    float w1 = param_blobs_[0]->cpu_data()[0];
    float theta_beta  = param_blobs_[1]->cpu_data()[0];
    
    int count = top[0]->count();
    gaussian_function_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
        (count, input_data, output_data,
         bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
         w1, theta_beta);

//    LOG(INFO)<<"w1, w2, alpha, beta, gamma= "<<param_blob_->cpu_data()[0]<<" "<<param_blob_->cpu_data()[1]<<" "<<
//        param_blob_->cpu_data()[2]<<" "<<param_blob_->cpu_data()[3]<<" "<<param_blob_->cpu_data()[4];
}
    
template <typename Dtype>
void PairwiseFunctionIntensityGaussianLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                        const vector<bool>& propagate_down,
                                                        const vector<Blob<Dtype>*>& bottom)
{
    Backward_cpu(top, propagate_down, bottom);
}
    
INSTANTIATE_LAYER_GPU_FUNCS(PairwiseFunctionIntensityGaussianLayer);
}  // namespace caffe
