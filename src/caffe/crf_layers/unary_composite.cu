#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/crf_layers/unary_composite_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "pixel_access.cu"

namespace caffe {
    
    
template <typename Dtype>
__global__ void unary_composite_kernel(const int nthreads, const Dtype* bottom_data, const Dtype * image_data,  Dtype* top_data, Dtype * mask_data, int N, int C, int H, int W, int C_image, Dtype u_potential, Dtype dis_cv)
{
    CUDA_KERNEL_LOOP(index, nthreads){
        const int w = index % W;
        const int h = (index/W) % H;
        const int n = index / W / H;
        
        int scribble_channel = -1;
        for(int c=0; c<C; c++)
        {
            int ci = C_image - C +c;
            Dtype d = get_gpu_pixel(image_data, N, C_image, H, W, n, ci, h, w);
            
            if((d + dis_cv) < 1e-5 && (d + dis_cv) > -1e-5)
            {
                scribble_channel = c;
                break;
            }
        }
        Dtype mask_value = (scribble_channel>-1)? 1.0 : 0.0;
        set_gpu_pixel(mask_data, N, 1, H, W, n, 0, h, w, mask_value);
        if(scribble_channel>-1)
        {
            for(int c=0; c<C; c++)
            {
                Dtype u_value = 0;//get_gpu_pixel(bottom_data, N, C, H, W, n, c, h, w);
                u_value = (c == scribble_channel)? u_value + u_potential : u_value - u_potential;
                set_gpu_pixel(top_data, N, C, H, W, n, c, h, w, u_value);
            }
        }
    }
}
    
template <typename Dtype>
void UnaryCompositeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top) {
    top[0]->CopyFrom(*bottom[0], false);// data is copied
    
    const Dtype * bottom_data = bottom[0]->gpu_data();
    const Dtype * image_data = bottom[1]->gpu_data();
    
    Dtype * top_data = top[0]->mutable_gpu_data();
    Dtype * mask_data = top[1]->mutable_gpu_data();
    int count = num_ * height_ * width_;
    Dtype user_potential = this->layer_param_.multi_stage_crf_param().user_interaction_potential();
//    LOG(INFO) << "user_interaction_potential "<< user_potential;
    Dtype dis_mean = this->layer_param_.multi_stage_crf_param().interaction_dis_mean();
    Dtype dis_std  = this->layer_param_.multi_stage_crf_param().interaction_dis_std();
    Dtype dis_cv = dis_mean/dis_std;
    
    unary_composite_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
    (count, bottom_data, image_data, top_data, mask_data, num_, unary_channels_, height_, width_, image_channels_, user_potential, dis_cv);
}
 
template <typename Dtype>
void UnaryCompositeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                              const vector<bool>& propagate_down,
                                              const vector<Blob<Dtype>*>& bottom)
{
    if(propagate_down[0]==false)return;
    bottom[0]->CopyFrom(*top[0], true);
}
INSTANTIATE_LAYER_GPU_FUNCS(UnaryCompositeLayer);


} // namespace caffe