#include "hip/hip_runtime.h"
/*!
 *  \brief     A helper class for {@link MultiStageMeanfieldLayer} class, which is the Caffe layer that implements the
 *             CRF-RNN described in the paper: Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             This class itself is not a proper Caffe layer although it behaves like one to some degree.
 *
 *  \authors   Sadeep Jayasumana, Bernardino Romera-Paredes, Shuai Zheng, Zhizhong Su.
 *  \version   1.0
 *  \date      2015
 *  \copyright Torr Vision Group, University of Oxford.
 *  \details   If you use this code, please consider citing the paper:
 *             Shuai Zheng, Sadeep Jayasumana, Bernardino Romera-Paredes, Vibhav Vineet, Zhizhong Su, Dalong Du,
 *             Chang Huang, Philip H. S. Torr. Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             For more information about CRF-RNN, please visit the project website http://crfasrnn.torr.vision.
 */
#include <vector>
#include <math.h>
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/loss_layer.hpp"
#include "caffe/layers/softmax_layer.hpp"
#include "caffe/crf_layers/message_passing_layer.hpp"
#include "caffe/crf_layers/pixel_access.hpp"
#include "pixel_access.cu"
namespace caffe {
    
template <typename Dtype>
__global__ void conv_kernel(const int nthreads, const Dtype* bottom, const Dtype* kernel, const Dtype* mask_data,Dtype* top, int N, int C, int H, int W, int neighN, bool user_interaction_constrain) {
    CUDA_KERNEL_LOOP(index, nthreads){
        const int w = index % W;
        const int h = (index/W) % H;
        const int c = (index / W / H) % C;
        const int n = index / W / H / C;
        
        int kernel_size = sqrt(double(neighN+1));
        int kr = (kernel_size -1)/2;
       
        int neighIdx=0;
        Dtype sum_value=0.0;
        bool interaction_exist =false;
        if(user_interaction_constrain){
         interaction_exist = get_gpu_pixel(mask_data, N, 1, H, W, n, 0, h, w)>0.0;
        }
        if(!(user_interaction_constrain && interaction_exist)){
            for(int i = -kr; i <= kr; i++)
            {
                for(int j = -kr; j <= kr; j++)
                {
                    if(i==0 && j==0) continue;
                    Dtype value = get_gpu_pixel(bottom, N, C, H, W, n, c, i+h, j+w);
                    Dtype weidht= get_gpu_pixel(kernel, N, neighN, H, W, n, neighIdx, h, w);
                    sum_value += value*weidht;
                    neighIdx++;
                }
            }
        }
        set_gpu_pixel(top, N, C, H, W, n, c, h, w, sum_value);
    }
}


template <typename Dtype>
__global__ void conv_gradient_to_input_kernel(const int nthreads, const Dtype* top_diff, const Dtype* kernel_data, const Dtype * mask_data, Dtype* bottom_diff, int N, int C, int H, int W, int neighN, bool user_interaction_constrain) {
    CUDA_KERNEL_LOOP(index, nthreads){
        const int w = index % W;
        const int h = (index/W) % H;
        const int c = (index / W / H) % C;
        const int n = index / W / H / C;
        
        int kernel_size = sqrt(double(neighN+1));
        int kr = (kernel_size -1)/2;

        int q_index=0;
        Dtype value_diff = 0.0;
        for(int i = -kr; i <= kr; i++)
        {
            for(int j = -kr; j <= kr; j++)
            {
                if(i==0 && j==0) continue;
                int nq_index = neighN-1 -q_index;
                Dtype weight_nq = get_gpu_pixel(kernel_data, N, neighN, H, W, n, nq_index, i+h, j+w);
                if(user_interaction_constrain &&
                   get_gpu_pixel(mask_data, N, 1, H, W, n, 0, i+h, j+w)){
                    weight_nq = 0;
                }
                Dtype t_diff_nq = get_gpu_pixel(top_diff, N, C, H, W, n, c, i+h, j+w);
                value_diff += weight_nq* t_diff_nq;
                q_index++;
            }
        }
        set_gpu_pixel(bottom_diff, N, C, H, W, n, c, h, w, value_diff);
    }
}
    
template <typename Dtype>
__global__ void conv_gradient_to_weight_kernel(const int nthreads, const Dtype* top_diff, const Dtype* bottom_data, const Dtype * mask_data, Dtype* kernel_diff, int N, int C, int H, int W, int neighN, bool user_interaction_constrain) {
    CUDA_KERNEL_LOOP(index, nthreads){
        const int w = index % W;
        const int h = (index/W) % H;
        const int c = (index / W / H) % neighN;
        const int n = index / W / H / neighN;
        
        if(user_interaction_constrain &&
           get_gpu_pixel(mask_data, N, 1, H, W, n, 0, h, w)){
            return;
        }
        int kernel_size = sqrt(double(neighN+1));
        int kr = (kernel_size -1)/2;
        
        int cN = (c >= kr*kernel_size + kr)? c+1 : c;
       
        int j = cN % kernel_size - kr;
        int i = cN / kernel_size - kr;
        Dtype k_diff = 0;
        for(int cIdx = 0; cIdx<C; cIdx++)
        {
            Dtype t_diff = get_gpu_pixel(top_diff, N, C, H, W, n, cIdx, h, w);
            Dtype value = get_gpu_pixel(bottom_data, N, C, H, W, n, cIdx, h+i, w+j);
            k_diff += value * t_diff;
        }
        //k_diff = k_diff / C;
        set_gpu_pixel(kernel_diff, N, neighN, H, W, n, c, h, w, k_diff);
    }
}

template <typename Dtype>
void MessagePassingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top)
{
    const Dtype * input_data  = bottom[0]->gpu_data();
    const Dtype * kernel_data = bottom[1]->gpu_data();
    const Dtype * mask_data   = (user_interaction_constrain_)? bottom[2]->gpu_data(): NULL;
    
    Dtype * output_data=top[0]->mutable_gpu_data();
    CHECK_EQ(bottom[0]->count(), top[0]->count())<<
        ("input image and output image shoud have the same size");
    CHECK(bottom[0]->height() == bottom[1]->height() && bottom[0]->width() == bottom[1]->width())<<
        ("input image and kernel shoud have the pixel number");

    int count = top[0]->count();
    conv_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
        (count, input_data, kernel_data, mask_data, output_data,
        bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
        bottom[1]->channels(), user_interaction_constrain_);
}

template <typename Dtype>
void MessagePassingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom)
{
    //LOG(INFO) << ("message pasing backward_gpu start.");
    const Dtype * top_diff = top[0]->gpu_diff();
    const Dtype * bottom_data = bottom[0]->gpu_data();
    Dtype * bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype * kernel_data = bottom[1]->gpu_data();
    Dtype * kernel_diff = bottom[1]->mutable_gpu_diff();
    const Dtype * mask_data = (user_interaction_constrain_)? bottom[2]->gpu_data(): NULL;
    
    int bottom_count = bottom[0]->count();
    conv_gradient_to_input_kernel<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>
        (bottom_count, top_diff, kernel_data, mask_data, bottom_diff,
         bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
         bottom[1]->channels(), user_interaction_constrain_);
   
    int kernel_count = bottom[1]->count();
    conv_gradient_to_weight_kernel<Dtype><<<CAFFE_GET_BLOCKS(kernel_count), CAFFE_CUDA_NUM_THREADS>>>
        (kernel_count, top_diff, bottom_data, mask_data, kernel_diff,
        bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
        bottom[1]->channels(), user_interaction_constrain_);
}

INSTANTIATE_LAYER_GPU_FUNCS(MessagePassingLayer);
}  // namespace caffe
