
#include <hip/hip_runtime.h>
/*!
 *  \brief     A helper class for {@link MultiStageMeanfieldLayer} class, which is the Caffe layer that implements the
 *             CRF-RNN described in the paper: Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             This class itself is not a proper Caffe layer although it behaves like one to some degree.
 *
 *  \authors   Sadeep Jayasumana, Bernardino Romera-Paredes, Shuai Zheng, Zhizhong Su.
 *  \version   1.0
 *  \date      2015
 *  \copyright Torr Vision Group, University of Oxford.
 *  \details   If you use this code, please consider citing the paper:
 *             Shuai Zheng, Sadeep Jayasumana, Bernardino Romera-Paredes, Vibhav Vineet, Zhizhong Su, Dalong Du,
 *             Chang Huang, Philip H. S. Torr. Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             For more information about CRF-RNN, please visit the project website http://crfasrnn.torr.vision.
 */

namespace caffe {

template <typename Dtype>
__device__ Dtype get_gpu_pixel(const Dtype* data, int C, int H, int W, int c, int h, int w)
{
    Dtype value=0;
    if( c>=0 && c<C && h>=0 && h<H && w>=0 && w<W)
    {
        value = data[H*W*c+h*W+w];
    }
    return value;
}

template <typename Dtype>
__device__ Dtype get_gpu_pixel(const Dtype* data, int N, int C, int H, int W, int n, int c, int h, int w)
{
    Dtype value=0;
    if( n>=0 && n<N && c>=0 && c<C && h>=0 && h<H && w>=0 && w<W)
    {
        value = data[n*C*H*W + c*H*W + h*W + w];
    }
    return value;
}

template <typename Dtype>
__device__ void set_gpu_pixel(Dtype* data, int C, int H, int W, int c, int h, int w, Dtype value)
{
    if( c>=0 && c<C && h>=0 && h<H && w>=0 && w<W)
    {
        data[H*W*c+h*W+w] = value;
    }
}

template <typename Dtype>
__device__ void set_gpu_pixel(Dtype* data, int N, int C, int H, int W, int n, int c, int h, int w, Dtype value)
{
    if(n>=0 && n<N && c>=0 && c<C && h>=0 && h<H && w>=0 && w<W)
    {
        data[n*C*H*W + c*H*W + h*W + w] = value;
    }
}
}  // namespace caffe
